
#include <hip/hip_runtime.h>
__global__  void somme( int taille, float * a, float * b, float *c  ){

     int index=threadIdx.x+blockDim.x*blockIdx.x;    
     if(index>=taille) return;
     c[index]=a[index]+b[index];
}


__global__  void prod( int taille, float * a, float  b, float *c  ){

     int index=threadIdx.x+blockDim.x*blockIdx.x;    
     if(index>=taille) return;
     c[index]=a[index]*b;
}